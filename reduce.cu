#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// 相邻配对
__global__ void reduceNeighbored(int *idata, int *odata, size_t n) {
    size_t tid = threadIdx.x;

    int *data = idata + blockIdx.x * blockDim.x;

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            data[tid] += data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)   odata[blockIdx.x] = data[0];
}

// 相邻配对 + 修改每个线程访问的数组元素
__global__ void reduceNeighbored(int *idata, int *odata, size_t n) {
    size_t tid = threadIdx.x;

    int *data = idata + blockIdx.x * blockDim.x;

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x>) {
            data[index] += data[index + stride];
        }
        __syncthreads();
    }

    if (tid == 0)   odata[blockIdx.x] = data[0];
}

// 交错配对
__global__ void reduceInterleaved(int *idata, int *odata, size_t n) {
    size_t tid = threadIdx.x;
    size_t idx = tid + blockDim.x * blockIdx.x;

    int *data = idata + blockDim.x * blockIdx.x;

    for (size_t stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)  {
            data[tid] += data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)  odata[blockIdx.x] = data[0];
}

// 展开循环   half blocks   grid.x / 2
__global__ void reduceUnrolling2(int *idata, int *odata, size_t n) {
    size_t tid = threadIdx.x;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x * 2;

    int *data = idata + blockDim.x * blockIdx.x * 2;
    if ((idx + blockDim.x) < n) data[tid] += data[tid + blockDim.x];
    __syncthreads();

    for (size_t stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            data[tid] += data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0)  odata[blockIdx.x] = data[0];
}


// 展开循环 & 线程束   half blocks   grid.x / 8
__global__ void reduceUnrollWarp8(int *idata, int *odata, size_t n) {
    size_t tid = threadIdx.x;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x * 8;

    int *data = idata + blockDim.x * blockIdx.x * 8;
    if ((idx + 7 * blockDim.x) < n) {
        int a0 = data[tid];
        int a1 = data[tid + blockDim.x];
        int a2 = data[tid + 2*blockDim.x];
        int a3 = data[tid + 3*blockDim.x];
        int a4 = data[tid + 4*blockDim.x];
        int a5 = data[tid + 5*blockDim.x];
        int a6 = data[tid + 6*blockDim.x];
        int a7 = data[tid + 7*blockDim.x];
        data[tid] = a0 + a1 + a2 + a3 + a4 + a5 + a6 + a7;
    }
    __syncthreads();

    for (size_t stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            data[tid] += data[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        volatile int *vmem = data;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if (tid == 0)  odata[blockIdx.x] = data[0];
}


int main(int argc, char **argv) {
    int dev = 0;
    hipSetDevice(dev);

    bool bResult = false;

    int size = 1<<24;
    int blockSize = 512;
    if (argc > 1) {
        blockSize = atoi(argv[1]);
    }
    dim3 block(blockSize, 1);
    dim3 grid((size + blockSize - 1) / blockSize);

    int *h_idata = (int *)malloc(size * sizeof(int));
    int *h_odata = (int *)malloc(grid.x * sizeof(int));
    int *tmp = (int *)malloc(size * sizeof(int));

    for (int i = 0; i < size; i++) {
        h_idata[i] = (int)(rand() & 0xFF);
    }
    memcpy(tmp, h_idata, size * sizeof(int));
    size_t istart, ielaps;
    int gpu_sum = 0;

    int *d_idata = NULL, *d_odata = NULL;
    hipMalloc((void **)&d_idata, size * sizeof(int));
    hipMalloc((void **)&d_odata, grid.x * sizeof(int));

    hipMemcpy(d_idata, h_idata, size * sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    istart = second();
    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    elaps = second() - istart;
}